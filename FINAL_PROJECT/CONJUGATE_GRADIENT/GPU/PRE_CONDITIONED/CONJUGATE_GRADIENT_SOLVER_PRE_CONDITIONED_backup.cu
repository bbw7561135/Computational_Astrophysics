#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void FPRINTF(FILE*, int N, double, double*);
double EVALUATE_ERROR(int, int, double*);

__global__ void INITIALIZE(int N, double dx, double* rho_even, double *rho_odd, double* field_even, double* field_odd, double* field_analytic, double* rho_analytic)
{
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + idx_y*N;
	int idx_eo = idx_x/2 + idx_y*(N/2);

	double x = idx_x*dx;
	double y = idx_y*dx;

	field_analytic[idx] = x*(1.-x)*y*(1.-y)*exp(x-y);
		
	if (((idx_x%2)+(idx_y%2))%2==0)
	{
		if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
		{
			field_even[idx_eo] = 0.0;
			rho_even[idx_eo] = (2.*x*(y-1)*(y-2.*x+x*y+2)*exp(x-y))*dx*dx;	// Notice that rho has been times by dx^2!!
		}
		else
		{
			field_even[idx_eo] = field_analytic[idx];
			rho_even[idx_eo] = 0.0;
		}
		rho_analytic[idx] = rho_even[idx_eo];	// Notice that rho has been times by dx^2!!
	}
	else
	{
		if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
		{
			field_odd[idx_eo] = 0.0;
			rho_odd[idx_eo] = (2.*x*(y-1)*(y-2.*x+x*y+2)*exp(x-y))*dx*dx;	// Notice that rho has been times by dx^2!!
		}
		else
		{
			field_odd[idx_eo] = field_analytic[idx];
			rho_odd[idx_eo] = 0.0;
		}
		rho_analytic[idx] = rho_odd[idx_eo];	// Notice that rho has been times by dx^2!!
	}
}

__global__ void EVALUATE_ERROR_BLOCK_EVEN(int N, double* rho, double* field_self, double* field_neighbor, double* error_block)
{
	extern __shared__ double sm[];
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + N/2*idx_y;
	int idx_sm = threadIdx.x + blockDim.x*threadIdx.y;

	if (idx_y!=0&&idx_y!=N-1)
	{
		if (idx_y%2==0)
		{
			if (idx_x!=0)
			{
				int L = idx - 1;
				int R = idx;
				int U = idx + N/2;
				int D = idx - N/2;
				sm[idx_sm] = pow((field_neighbor[L]+field_neighbor[R]+field_neighbor[U]+field_neighbor[D]-4.*field_self[idx])-rho[idx], 2.);
			}
			else
				sm[idx_sm] = 0.0;
		}
		else
		{
			if (idx_x!=N/2-1)
			{
				int L = idx;
				int R = idx + 1;
				int U = idx + N/2;
				int D = idx - N/2;
				sm[idx_sm] = pow((field_neighbor[L]+field_neighbor[R]+field_neighbor[U]+field_neighbor[D]-4.*field_self[idx])-rho[idx], 2.);
			}
			else
				sm[idx_sm] = 0.0;
		}
	}
	else
		sm[idx_sm] = 0.0;
	__syncthreads();

	for (int shift=blockDim.x*blockDim.y/2; shift>0; shift/=2)
	{
		if (idx_sm<shift)
			sm[idx_sm] += sm[idx_sm+shift];
		__syncthreads();
	}
	if (idx_sm==0)
		error_block[blockIdx.x+blockIdx.y*gridDim.x] = sm[0];
//	printf("%d\t%.4f\n", blockIdx.x+gridDim.x*blockIdx.y, sm[0]);
}

__global__ void EVALUATE_ERROR_BLOCK_ODD(int N, double* rho, double* field_self, double* field_neighbor, double* error_block)
{
	extern __shared__ double sm[];
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + N/2*idx_y;
	int idx_sm = threadIdx.x + blockDim.x*threadIdx.y;

	if (idx_y!=0&&idx_y!=N-1)
	{
		if (idx_y%2==0)
		{
			if (idx_x!=N/2-1)
			{
				int L = idx;
				int R = idx + 1;
				int U = idx + N/2;
				int D = idx - N/2;
				sm[idx_sm] = pow((field_neighbor[L]+field_neighbor[R]+field_neighbor[U]+field_neighbor[D]-4.*field_self[idx])-rho[idx], 2.);
			}
			else
				sm[idx_sm] = 0.0;
		}
		else
		{
			if (idx_x!=0)
			{
				int L = idx - 1;
				int R = idx;
				int U = idx + N/2;
				int D = idx - N/2;
				sm[idx_sm] = pow((field_neighbor[L]+field_neighbor[R]+field_neighbor[U]+field_neighbor[D]-4.*field_self[idx])-rho[idx], 2.);
			}
			else
				sm[idx_sm] = 0.0;
		}
	}
	else
		sm[idx_sm] = 0.0;
	__syncthreads();

	for (int shift=blockDim.x*blockDim.y/2; shift>0; shift/=2)
	{
		if (idx_sm<shift)
			sm[idx_sm] += sm[idx_sm+shift];
		__syncthreads();
	}
	if (idx_sm==0)
		error_block[blockIdx.x+blockIdx.y*gridDim.x] = sm[0];
//	printf("%d\t%.4f\n", blockIdx.x+gridDim.x*blockIdx.y, sm[0]);
}

__global__ void LAPLACIAN_EVEN(int N, double dx, double photon_mass, double* p_self, double* p_neighbor, double* A_p)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N/2*idx_y;

	if (idx_y!=0&&idx_y!=N-1)
	{
		if (idx_y%2==0)
		{
			if (idx_x!=0)
			{
				int L = idx - 1;
				int R = idx;
				int U = idx + N/2;
				int D = idx - N/2;
				A_p[idx] = (p_neighbor[L]+p_neighbor[R]+p_neighbor[U]+p_neighbor[D]-(4.-pow(photon_mass*dx,2.))*p_self[idx]);
			}
			else
				A_p[idx] = 0.0;
		}
		else
		{
			if (idx_x!=N/2-1)
			{
				int L = idx;
				int R = idx + 1;
				int U = idx + N/2;
				int D = idx - N/2;
				A_p[idx] = (p_neighbor[L]+p_neighbor[R]+p_neighbor[U]+p_neighbor[D]-(4.-pow(photon_mass*dx,2.))*p_self[idx]);
			}
			else
				A_p[idx] = 0.0;
		}
	}
	else
		A_p[idx] = 0.0;
}

__global__ void LAPLACIAN_ODD(int N, double dx, double photon_mass, double* p_self, double* p_neighbor, double* A_p)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N/2*idx_y;

	if (idx_y!=0&&idx_y!=N-1)
	{
		if (idx_y%2==0)
		{
			if (idx_x!=N/2-1)
			{
				int L = idx;
				int R = idx + 1;
				int U = idx + N/2;
				int D = idx - N/2;
				A_p[idx] = (p_neighbor[L]+p_neighbor[R]+p_neighbor[U]+p_neighbor[D]-(4.-pow(photon_mass*dx,2.))*p_self[idx]);
			}
			else
				A_p[idx] = 0.0;
		}
		else
		{
			if (idx_x!=0)
			{
				int L = idx - 1;
				int R = idx;
				int U = idx + N/2;
				int D = idx - N/2;
				A_p[idx] = (p_neighbor[L]+p_neighbor[R]+p_neighbor[U]+p_neighbor[D]-(4.-pow(photon_mass*dx,2.))*p_self[idx]);
			}
			else
				A_p[idx] = 0.0;
		}
	}
	else
		A_p[idx] = 0.0;
}

__global__ void DAXPY(int N, double c, double* A, double* B)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;

	A[idx] = c*A[idx] + B[idx];
}

int main(void)
{
	int N, N_block, display_interval, tpb_x, tpb_y, bpg_x, bpg_y, shift, shift_block;
	float preparation_time, computation_time, total_time;
	double photon_mass, omega, dx, criteria;
	double alpha, beta;
	long iter, iter_max;
	double *field, *rho, *r, *p, *A_p, *field_analytic, *rho_analytic, *error_block;
	size_t size_lattice, size_sm;
	hipEvent_t start, stop;
	FILE* output_field, *output_rho;
	printf("Solve the Poission problem using CG by GPU with pre-conditioned.\n\n");
	printf("Enter the latttice size (N,N) (N must be divisible by 2).");
	scanf("%d", &N);
	if (N%2!=0)
	{
		printf("N is not divisible by 2! Exit!\n");
		exit(1);
	}
	printf("The lattice size is (%d,%d).\n", N, N);
	printf("Set the photon mass.\n");
	scanf("%lf", &photon_mass);
	printf("The photon mass is %.4e .\n", photon_mass);
	printf("Set the value of omega.\n");
	scanf("%lf", &omega);
	printf("The value of omega is %.4e .\n", omega);
	printf("Set the maximum iteration times.\n");
	scanf("%ld", &iter_max);
	printf("The maximum iteration times is %ld .\n", iter_max);
	printf("Set the stopping criteria.\n");
	scanf("%lf", &criteria);
	printf("The stopping criteria is %.4e .\n", criteria);
	printf("Set the display interval during iterations.\n");
	scanf("%d", &display_interval);
	printf("The display interval is set to be %d .\n", display_interval);
	printf("Set the GPU threads per block (tx,ty). (N/2 must be divisible by tx and N must be divisible by ty)\n");
	scanf("%d %d", &tpb_x, &tpb_y);
	if ((N/2)%tpb_x!=0)
	{
		printf("N/2 is not divisible by tx! Exit!\n");
		return EXIT_FAILURE;
	}
	else if (N%tpb_y!=0)
	{
		printf("N is not divisible by ty! Exit!\n");
		return EXIT_FAILURE;
	}
	else
	{
		printf("Threads per block for GPU is (%d,%d) .\n", tpb_x, tpb_y);
		printf("The block per grid will be set automatically.");
		bpg_x = (N/2)/tpb_x;
		bpg_y = N/tpb_y;
		printf("Blocks per grid for GPU is (%d,%d) .\n", bpg_x, bpg_y);
	}
	printf("\n");

	printf("Start Preparation...\n");
	N_block = 2*bpg_x*bpg_y;
	shift = (N/2)*N;
	shift_block = bpg_x*bpg_y;
	dx = 1./(N-1);	
	size_lattice = N*N*sizeof(double);
	size_sm = tpb_x*tpb_y*sizeof(double);
	output_field = fopen("analytical_field_distribution_CG_pre.txt","w");
	output_rho = fopen("charge_distribution_CG_pre.txt","w");

	hipSetDevice(0);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 tpb(tpb_x,tpb_y);
	dim3 bpg(bpg_x,bpg_y);
	dim3 bpg_init(2*bpg_x,bpg_y);
	hipblasMath_t mode = HIPBLAS_TENSOR_OP_MATH;
    hipblasPointerMode_t mode_pt = HIPBLAS_POINTER_MODE_HOST;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	hipblasSetMathMode(handle, mode);
    hipblasSetPointerMode(handle, mode_pt);

	hipEventRecord(start,0);
	hipMallocManaged(&field, size_lattice);
	hipMallocManaged(&r, size_lattice);
	hipMallocManaged(&p, size_lattice);
	hipMallocManaged(&A_p, size_lattice);
	hipMallocManaged(&rho, size_lattice);
	hipMallocManaged(&error_block, N_block*sizeof(double));
	hipMallocManaged(&field_analytic, 2*size_lattice);
	hipMallocManaged(&rho_analytic, 2*size_lattice);

	INITIALIZE<<<bpg_init,tpb>>>(N, dx, rho, rho+shift, field, field+shift, field_analytic, rho_analytic);
	EVALUATE_ERROR_BLOCK_EVEN<<<bpg,tpb,size_sm>>>(N, rho, field, field+shift, error_block);
	EVALUATE_ERROR_BLOCK_ODD<<<bpg,tpb,size_sm>>>(N, rho+shift, field+shift, field, error_block+shift_block);
	double norm;
	hipblasDdot(handle, N*N, rho, 1, rho, 1, &norm);
	norm = sqrt(norm);
	
	hipDeviceSynchronize();
	hipMemcpy(r, rho, size_lattice, hipMemcpyDeviceToDevice);
	hipMemcpy(p, rho, size_lattice, hipMemcpyDeviceToDevice);
	
	FPRINTF(output_field, N, 1., field_analytic);
	FPRINTF(output_rho, N, pow(dx,-2.), rho_analytic);
	hipEventRecord(start,0);

	printf("Preparation ends.\n");
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&preparation_time, start, stop);
	printf("Total preparation time is %.4f ms.\n\n", preparation_time);

	hipEventRecord(start,0);	
	double error = EVALUATE_ERROR(N, N_block, error_block); 
	double temp;

	printf("Starts computation with error = %.8e...\n", sqrt(error)/norm);
	iter = 0;
	
	while (sqrt(error)/norm>criteria&&iter<iter_max)
	{
		LAPLACIAN_EVEN<<<bpg,tpb>>>(N, dx, photon_mass, p, p+shift, A_p);
		LAPLACIAN_ODD<<<bpg,tpb>>>(N, dx, photon_mass, p+shift, p, A_p+shift);
		hipblasDdot(handle, N*N, p, 1, A_p, 1, &temp);
		alpha = error/temp;
		temp = -alpha;
		hipblasDaxpy(handle, N*N, &temp, A_p, 1, r, 1);
		hipblasDaxpy(handle, N*N, &alpha, p, 1, field, 1);
		hipblasDdot(handle, N*N, r, 1, r, 1, &temp);
		beta = temp/error;
//		printf("%.4f\t%.4f\n", alpha, beta);
		DAXPY<<<bpg_init,tpb>>>(N, beta, p, r);
		error = temp;
		iter += 1;
		if (iter%display_interval==0)
			printf("Iteration = %ld , error = %.8e .\n", iter, sqrt(error)/norm);
	}
  
	output_field = fopen("simulated_field_distribution_GPU_CG.txt","w");
	FPRINTF(output_field, N, 1., field);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computation_time, start, stop);
	printf("Computation time is %.4f ms.\n", computation_time);
	total_time = preparation_time + computation_time;
	printf("Total iteration is %ld ; total time is %.4f ms.\n", iter, total_time);

	hipFree(field);
	hipFree(r);
	hipFree(p);
	hipFree(A_p);
	hipFree(field_analytic);
	hipFree(rho_analytic);
	hipFree(rho);
	hipFree(error_block);
	hipblasDestroy(handle);
	fclose(output_field);
	fclose(output_rho);
	return EXIT_SUCCESS;
}

double EVALUATE_ERROR(int N, int N_block, double* error_block)
{
	double error = 0.0;
	for (int i=0; i<N_block; i++)
		error += error_block[i];
	return error;
}

void FPRINTF(FILE *output_file, int N, double scale, double *array)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
			fprintf(output_file, "%.8e\t", scale*array[i+j*N]);
		fprintf(output_file, "\n");
	}
}
