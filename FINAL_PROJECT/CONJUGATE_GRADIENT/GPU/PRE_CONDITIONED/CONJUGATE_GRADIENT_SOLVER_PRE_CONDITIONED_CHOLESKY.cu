/* Use Cholesky pre-condition */ 

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void FPRINTF(FILE*, int N, double, double*);
void PRE_CONDITION_SSOR(int N, double, double, double, double*, double*);
double EVALUATE_ERROR(int, int, double*);
extern void dpotrf_(char*, int*, double*, int*, int*);

__global__ void INITIALIZE(int N, double dx, double photon_mass, double* rho, double* field, double* field_analytic, double *A)
{
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + idx_y*N;
	int row = N*N;

	double x = idx_x*dx;
	double y = idx_y*dx;

	field_analytic[idx] = x*(1.-x)*y*(1.-y)*exp(x-y);
		
	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		field[idx] = 0.0;
		rho[idx] = -(2.*x*(y-1)*(y-2.*x+x*y+2)*exp(x-y))*dx*dx;	// Notice that rho has been times by dx^2!!

        if (idx_x>1)
            A[(idx-1)*row + idx] = -1.;
        if (idx_x<N-2)
            A[(idx+1)*row + idx] = -1.;
        if (idx_y>1)
            A[(idx-N)*row + idx] = -1.;
        if (idx_y<N-2)
            A[(idx+N)*row + idx] = -1.;
        A[idx*row + idx] = 4. - pow(photon_mass*dx,2.);
	}
	else
	{
		field[idx] = field_analytic[idx];
		rho[idx] = 0.0;

        A[idx*row + idx] = 1.;
	}
}

__global__ void EVALUATE_ERROR_BLOCK(int N, double* rho, double* field, double* error_block)
{
	extern __shared__ double sm[];
	int idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	int idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = idx_x + N*idx_y;
	int idx_sm = threadIdx.x + blockDim.x*threadIdx.y;

	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		int L = idx_x-1 + idx_y*N;
		int R = idx_x+1 + idx_y*N;
		int U = idx_x + (idx_y+1)*N;
		int D = idx_x + (idx_y-1)*N;
		sm[idx_sm] = pow((4.*field[idx] - field[L] - field[R] - field[U] - field[D])-rho[idx], 2.);
	}
	else
		sm[idx_sm] = 0.0;
	__syncthreads();

	for (int shift=blockDim.x*blockDim.y/2; shift>0; shift/=2)
	{
		if (idx_sm<shift)
			sm[idx_sm] += sm[idx_sm+shift];
		__syncthreads();
	}
	if (idx_sm==0)
		error_block[blockIdx.x+blockIdx.y*gridDim.x] = sm[0];
//	printf("%d\t%.4f\n", blockIdx.x+gridDim.x*blockIdx.y, sm[0]);
}

__global__ void LAPLACIAN(int N, double dx, double photon_mass, double* p, double* A_p)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;
	
	if (idx_x!=0&&idx_x!=N-1&&idx_y!=0&&idx_y!=N-1)
	{
		int L = idx_x-1 + idx_y*N;
		int R = idx_x+1 + idx_y*N;
		int U = idx_x + (idx_y+1)*N;
		int D = idx_x + (idx_y-1)*N;

		A_p[idx] = ((4.-pow(photon_mass*dx,2.))*p[idx]-p[L]-p[R]-p[U]-p[D]);
//		printf("%d\t%.4f\n", idx, A_p[idx]);
	}
	else
		A_p[idx] = 0.0;
}

__global__ void DAXPY(int N, double c, double *A, double *B)
{
	int idx_x = threadIdx.x + blockDim.x*blockIdx.x;
	int idx_y = threadIdx.y + blockDim.y*blockIdx.y;
	int idx = idx_x + N*idx_y;

	A[idx] = c*A[idx] + B[idx];
}

int main(void)
{
	int N, N_block, display_interval, tpb_x, tpb_y, bpg_x, bpg_y;
	float preparation_time, computation_time, total_time;
	double photon_mass, dx, criteria;
	double alpha, beta;
	long iter, iter_max;
	double *field, *rho, *A, *r, *r_prime, *p, *A_p, *field_analytic, *error_block;
	size_t size_lattice, size_sm;
	hipEvent_t start, stop;
	FILE* output_field, *output_rho;
	printf("Solve the Poission problem using CG with SSOR precondition by GPU.\n\n");
	printf("Enter the latttice size (N,N) .");
	scanf("%d", &N);
	printf("The lattice size is (%d,%d).\n", N, N);
	printf("Set the photon mass.\n");
	scanf("%lf", &photon_mass);
	printf("The photon mass is %.4e .\n", photon_mass);
	printf("Set the maximum iteration times.\n");
	scanf("%ld", &iter_max);
	printf("The maximum iteration times is %ld .\n", iter_max);
	printf("Set the stopping criteria.\n");
	scanf("%lf", &criteria);
	printf("The stopping criteria is %.4e .\n", criteria);
	printf("Set the display interval during iterations.\n");
	scanf("%d", &display_interval);
	printf("The display interval is set to be %d .\n", display_interval);
	printf("Set the GPU threads per block (tx,ty). (N must be divisible by tx and N must be divisible by ty)\n");
	scanf("%d %d", &tpb_x, &tpb_y);
	if (N%tpb_x!=0)
	{
		printf("N is not divisible by tx! Exit!\n");
		return EXIT_FAILURE;
	}
	else if (N%tpb_y!=0)
	{
		printf("N is not divisible by ty! Exit!\n");
		return EXIT_FAILURE;
	}
	else
	{
		printf("Threads per block for GPU is (%d,%d) .\n", tpb_x, tpb_y);
		printf("The block per grid will be set automatically.");
		bpg_x = N/tpb_x;
		bpg_y = N/tpb_y;
		printf("Blocks per grid for GPU is (%d,%d) .\n", bpg_x, bpg_y);
	}
	printf("\n");

	printf("Start Preparation...\n");
	dx = 1./(N-1);	
	N_block = bpg_x*bpg_y;
	size_lattice = N*N*sizeof(double);
	size_sm = tpb_x*tpb_y*sizeof(double);
	output_field = fopen("analytical_field_distribution_CG_precondition_CHOLESKY.txt","w");
	output_rho = fopen("charge_distribution_CG_precondition_CHOLESKY.txt","w");

	hipSetDevice(0);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 tpb(tpb_x,tpb_y);
	dim3 bpg(bpg_x,bpg_y);
	hipblasMath_t mode = HIPBLAS_TENSOR_OP_MATH;
    hipblasPointerMode_t mode_pt = HIPBLAS_POINTER_MODE_HOST;
	hipblasHandle_t handle;

	hipblasCreate(&handle);
	hipblasSetMathMode(handle, mode);
    hipblasSetPointerMode(handle, mode_pt);

	hipEventRecord(start,0);
	hipMallocManaged(&A, N*N*N*N*sizeof(double));
	hipMallocManaged(&field, size_lattice);
	hipMallocManaged(&r, size_lattice);
	hipMallocManaged(&r_prime, size_lattice);
	hipMallocManaged(&p, size_lattice);
	hipMallocManaged(&A_p, size_lattice);
	hipMallocManaged(&field_analytic, size_lattice);
	hipMallocManaged(&rho, size_lattice);
	hipMallocManaged(&error_block, N_block*sizeof(double));
	hipMemset(A, 0, N*N*N*N*sizeof(double));

	INITIALIZE<<<bpg,tpb>>>(N, dx, photon_mass, rho, field, field_analytic, A);
	EVALUATE_ERROR_BLOCK<<<bpg,tpb,size_sm>>>(N, rho, field, error_block);

//	FILE* output_test = fopen("Matrix_A.txt","w");
//	FPRINTF(output_test, N*N, 1., A);

//    /* use the lapack function */
	char type = 'L';
	int row = N*N;
	int status;
    dpotrf_(&type, &row, A, &row, &status);	
	hipFree(A);
//
//	double norm;
//	cublasDdot(handle, N*N, rho, 1, rho, 1, &norm);
//	norm = sqrt(norm);
//	
//	cudaDeviceSynchronize();
//	cudaMemcpy(r, rho, size_lattice, cudaMemcpyDeviceToDevice);
//	
//	FPRINTF(output_field, N, 1., field_analytic);
//	FPRINTF(output_rho, N, -pow(dx,-2.), rho);
//	cudaEventRecord(start,0);
//
//	printf("Preparation ends.\n");
//	cudaEventRecord(stop,0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&preparation_time, start, stop);
//	printf("Total preparation time is %.4f ms.\n\n", preparation_time);
//
//	cudaEventRecord(start,0);	
//	double error = EVALUATE_ERROR(N, N_block, error_block); 
//	double temp;
//
//	printf("Starts computation with error = %.8e...\n", sqrt(error)/norm);
//	iter = 0;
//	PRE_CONDITION_SSOR(N, dx, photon_mass, omega, r, r_prime);
//
////	for (int i=0; i<N*N; i++)
////		printf("%.4f\n", r_prime[i]);
//
//	cudaMemcpy(p, r_prime, size_lattice, cudaMemcpyDeviceToDevice);
//	
//	while (sqrt(error)/norm>criteria&&iter<iter_max)
//	{
//		LAPLACIAN<<<bpg,tpb>>>(N, dx, photon_mass, p, A_p);
//		cublasDdot(handle, N*N, p, 1, A_p, 1, &temp);
//		cublasDdot(handle, N*N, r, 1, r_prime, 1, &beta);
//		alpha = beta/temp;
//		temp = -alpha;
//		cublasDaxpy(handle, N*N, &temp, A_p, 1, r, 1);
//		cublasDaxpy(handle, N*N, &alpha, p, 1, field, 1);
//		cudaDeviceSynchronize();
//		PRE_CONDITION_SSOR(N, dx, photon_mass, omega, r, r_prime);
//		cublasDdot(handle, N*N, r, 1, r_prime, 1, &temp);
//		beta = temp/beta;
////		printf("%.4f\t%.4f\n", alpha, beta);
//		DAXPY<<<bpg,tpb>>>(N, beta, p, r_prime);
//		cublasDdot(handle, N*N, r, 1, r, 1, &error);
//		iter += 1;
//		if (iter%display_interval==0)
//			printf("Iteration = %ld , error = %.8e .\n", iter, sqrt(error)/norm);
//	}
//  
//	output_field = fopen("simulated_field_distribution_GPU_CG_precondition_SSOR.txt","w");
//	FPRINTF(output_field, N, 1., field);
//	cudaEventRecord(stop,0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&computation_time, start, stop);
//	printf("Computation time is %.4f ms.\n", computation_time);
//	total_time = preparation_time + computation_time;
//	printf("Total iteration is %ld ; total time is %.4f ms.\n", iter, total_time);

	hipFree(field);
	hipFree(r);
	hipFree(r_prime);
	hipFree(p);
	hipFree(A_p);
	hipFree(field_analytic);
	hipFree(rho);
	hipFree(error_block);
	hipblasDestroy(handle);
	fclose(output_field);
	fclose(output_rho);
	return EXIT_SUCCESS;
}

double EVALUATE_ERROR(int N, int N_block, double* error_block)
{
	double error = 0.0;
	for (int i=0; i<N_block; i++)
		error += error_block[i];
	return error;
}

void PRE_CONDITION_SSOR(int N, double dx, double photon_mass, double omega, double* r, double* r_prime)
{
    double *temp = (double*)calloc(N*N, sizeof(double));
    for (int idx=0; idx<N*N; idx++)
    {
        int idx_x = idx%N;
        int idx_y = idx/N;
        if ( idx_x!=0 && idx_x!=N-1 && idx_y!=0 && idx_y!=N-1 )
		{
			if (idx_x>1&&idx_y>1)
				temp[idx] = -omega*((2.-omega)*r[idx]-(temp[idx-1]+temp[idx-N]))/(4.-pow(photon_mass*dx,2.));
			else if (idx_x>1)
				temp[idx] = -omega*((2.-omega)*r[idx]-temp[idx-1])/(4.-pow(photon_mass*dx,2.));
			else if (idx_y>1)
				temp[idx] = -omega*((2.-omega)*r[idx]-temp[idx-N])/(4.-pow(photon_mass*dx,2.));
			else
				temp[idx] = -omega*(2.-omega)*r[idx]/(4.-pow(photon_mass*dx,2.));
		}
        else
            temp[idx] = omega*(2.-omega)*r[idx];
//      printf("temp[%d]\t%.8f\n", idx, temp[idx]);
    }                                                                  
    for (int idx=N*N-1; idx>=0; idx--)
    {
        int idx_x = idx%N;
        int idx_y = idx/N;
        if ( idx_x!=0 && idx_x!=N-1 && idx_y!=0 && idx_y!=N-1 )
        {
            temp[idx] *= pow(photon_mass*dx,2.) - 4.;
			if (idx_x<N-2&&idx_y<N-2)
				r_prime[idx] = -(temp[idx]-omega*(r_prime[idx+1]+r_prime[idx+N]))/(4.-pow(photon_mass*dx,2.));
			else if (idx_x<N-2)
				r_prime[idx] = -(temp[idx]-omega*r_prime[idx+1])/(4.-pow(photon_mass*dx,2.));
			else if (idx_y<N-2)
				r_prime[idx] = -(temp[idx]-omega*r_prime[idx+N])/(4.-pow(photon_mass*dx,2.));
			else
				r_prime[idx] = -temp[idx]/(4.-pow(photon_mass*dx,2.));
        }
        else
            r_prime[idx] = temp[idx];
    }
    free(temp);
}

void FPRINTF(FILE *output_file, int N, double scale, double *array)
{
	for (int j=0; j<N; j++)
	{
		for (int i=0; i<N; i++)
			fprintf(output_file, "%.4f\t", scale*array[i+j*N]);
//			fprintf(output_file, "%.8e\t", scale*array[i+j*N]);
		fprintf(output_file, "\n");
	}
}
