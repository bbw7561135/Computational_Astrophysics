/* Matrices are represented in column-major format. */
/* Matrix product calcualted by cublasgemm seems not precise enough, so A^HA is computed by self-defined function.*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define t_B 32					// threads per block
#define b_G 1024				// blocks per grid
#define amplitude 10.0			// scale the matrix and vector elements
#define IDX2C(i,j,ld) (j*ld+i)	// mapping of memory location

#define USE_CURAND false		// use the curand library to generate A and |b> randomly.
#define DO_CPU_CHECK true		// Do CG on CPU

void MHM_PRODUCT(int, hipDoubleComplex *, hipDoubleComplex *, hipDoubleComplex *);

#if (USE_CURAND==true)
int dimension;
__global__ void SET_RNG(long seed, curandState *state)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	curand_init(seed, index, 0, &state[index]);
}

__global__ void FILLING_WITH_RNG(int N, cuDoubleComplex *A, curandState *state)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int shift = blockDim.x*gridDim.x;
	curandState local = state[index];
	while (index<N)
	{
		A[index].x = amplitude*curand_uniform(&local);
		A[index].y = amplitude*curand_uniform(&local);
		index += shift;
	}
	state[threadIdx.x + blockIdx.x*blockDim.x] = local;
}
#else
#define dimension 5	
#endif

#if (DO_CPU_CHECK==true)
void VV_ADDITION(int, hipDoubleComplex *, hipDoubleComplex *, hipDoubleComplex *, hipDoubleComplex *, hipDoubleComplex *);
hipDoubleComplex VV_PRODUCT(int, hipDoubleComplex *, hipDoubleComplex *);
void MV_PRODUCT(char, int, hipDoubleComplex *, hipDoubleComplex *, hipDoubleComplex *);
#endif

int main(void)
{
	puts("Solve the linear system A|x>=|b> by conjugate gradient on GPU.\n");
#if (USE_CURAND==true)
	puts("Set the dimension for the vector space.");
	scanf("%d", &dimension);
	printf("The dimension for the vector is %d .\n\n", dimension);
#endif
	char print_out;
	int iter = 0;
	float time_GPU, time_CPU;
	double error, criteria, norm;
	hipDoubleComplex alpha, beta, lambda_k, u_k;
	hipDoubleComplex *A, *b, *temp_matrix;
	hipDoubleComplex *A_dev, *b_dev, *x_k_dev, *r_k_dev, *p_k_dev;
	hipDoubleComplex *x_k_host = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex)*dimension);
	hipEvent_t start, stop;

	A = (hipDoubleComplex *)malloc(dimension*dimension*sizeof(hipDoubleComplex));
	temp_matrix = (hipDoubleComplex *)malloc(dimension*dimension*sizeof(hipDoubleComplex));
	b = (hipDoubleComplex *)malloc(dimension*sizeof(hipDoubleComplex));
	hipMalloc((void **)&A_dev, sizeof(hipDoubleComplex)*dimension*dimension);
	hipMalloc((void **)&b_dev, sizeof(hipDoubleComplex)*dimension);
	hipMalloc((void **)&x_k_dev, sizeof(hipDoubleComplex)*dimension);
	hipMalloc((void **)&r_k_dev, sizeof(hipDoubleComplex)*dimension);
	hipMalloc((void **)&p_k_dev, sizeof(hipDoubleComplex)*dimension);
	hipEventCreate(&start);
	hipEventCreate(&stop);

#if (USE_CURAND==true)
	long seed = 684128862;
	curandState *state_dev;
	cudaMalloc((void **)&state_dev, sizeof(curandState)*t_B*b_G);
	SET_RNG<<<b_G, t_B>>>(seed, state_dev);
	FILLING_WITH_RNG<<<b_G, t_B>>>(dimension*dimension, A_dev, state_dev);
	FILLING_WITH_RNG<<<b_G, t_B>>>(dimension, b_dev, state_dev);
	cublasGetMatrix(dimension, dimension, sizeof(cuDoubleComplex), A_dev, dimension, A, dimension);
	cublasGetVector(dimension, sizeof(cuDoubleComplex), b_dev, 1, b, 1);
	cudaFree(state_dev);
	scanf("%c", &print_out);
#else
	int index;
	double A_real[dimension][dimension] = { {1.0, 0.5, 0.0, 0.2645, 0.0}, 
											{1.0, 0.33, 2.123, 0.0, 0.001},
											{0.0, 0.0, 0.215, 0.0, 0.0},
											{0.249, 0.0, 0.0131, 0.013, 1.0},
											{0.0, 0.123, 0.0127, 0.0, 0.011} };
	double A_imag[dimension][dimension] = { {0., 0.0, 0.0, 0.0, 0.0}, 
											{0.99, 0.0, 1.0, 0.0, 0.0},
											{0.0, 1.0, 0.0, 0.0, 0.0},
											{0.0, 0.0, 0.0, 0.0, -8.97},
											{0.0, 0.0, 0.0, 0.0, 0.0} };
	
	double b_real[dimension] = {1.001, 1.0, 0.0 ,0.0, 0.0};
	double b_imag[dimension] = {0.0, 0.877, 0.0 ,0.0, 0.0};

	for (int j=0; j<dimension; j++)
	{
		b[j] = make_hipDoubleComplex(b_real[j], b_imag[j]);
		x_k_host[j] = make_hipDoubleComplex(0.0, 0.0);
		for (int i=0; i<dimension; i++)
		{
			index = IDX2C(i,j,dimension);
			A[index] = make_hipDoubleComplex(A_real[j][i], A_imag[j][i]);
		}
	}
#endif
	
	hipblasMath_t mode = HIPBLAS_TENSOR_OP_MATH;
	hipblasPointerMode_t mode_pt = HIPBLAS_POINTER_MODE_HOST;
	hipblasHandle_t handle;

	puts("Print out the matrix A nd vector |b>? (y/n)");
	scanf("%c", &print_out);
	if (print_out=='y')
	{
		puts("Matrix A is:");
		for (int j=0; j<dimension; j++)
		{
			for (int i=0; i<dimension; i++)
				printf("%.6f%+.6fI\t", A[IDX2C(j,i,dimension)].x, A[IDX2C(j,i,dimension)].y);
			printf("\n");
		}
		printf("\n");
		puts("vector |b> is:");
		for (int i=0; i<dimension; i++)
			printf("%.6f%+.6fI\n", b[i].x, b[i].y);
		printf("\n");
	}
	else if (print_out!='n')
	{
		puts("Wrong input! Exit!");
		EXIT_FAILURE;
	}
	
	puts("Set the stopping criteria.");
	scanf("%lf", &criteria);
	printf("The stopping criteria is %.4e .\n", criteria);
	
	// generate A^HA
	MHM_PRODUCT(dimension, A, A, temp_matrix);
	//
	puts("Start conjugate gradient on GPU...");

	hipEventRecord(start, 0);
	hipblasCreate(&handle);
	hipblasSetMathMode(handle, mode);
	hipblasSetPointerMode(handle, mode_pt);

	hipblasSetMatrix(dimension, dimension, sizeof(hipDoubleComplex), A, dimension, A_dev, dimension);
	hipblasSetVector(dimension, sizeof(hipDoubleComplex), b, 1, b_dev, 1);
	hipblasSetVector(dimension, sizeof(hipDoubleComplex), x_k_host, 1, x_k_dev, 1);
	alpha = make_hipDoubleComplex(1.0, 0.0);
	beta = make_hipDoubleComplex(0.0, 0.0);

	//convert |b> to A^H|b>
	hipblasZgemv(handle, HIPBLAS_OP_C, dimension, dimension, &alpha, A_dev, dimension, b_dev, 1, &beta, b_dev, 1);
	//conver A to A^HA
//	cublasZgemm(handle, CUBLAS_OP_C, CUBLAS_OP_N, dimension, dimension, dimension, &alpha, A_dev, dimension, A_dev, dimension, &beta, A_dev, dimension);
	hipblasSetMatrix(dimension, dimension, sizeof(hipDoubleComplex), temp_matrix, dimension, A_dev, dimension);

	hipblasDznrm2(handle, dimension, b_dev, 1, &norm);
	hipMemcpy(r_k_dev, b_dev, dimension*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	hipMemcpy(p_k_dev, r_k_dev, dimension*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	error = 1.0;

	while (error>criteria)
	{
		// calculate A|p_k>
		hipblasZgemv(handle, HIPBLAS_OP_N, dimension, dimension, &alpha, A_dev, dimension, p_k_dev, 1, &beta, b_dev, 1);	//use b_dev to save A|p_k>
		// calculate lambda_k
		hipblasDznrm2(handle, dimension, r_k_dev, 1, &error);	
		hipblasZdotc(handle, dimension, p_k_dev, 1, b_dev, 1, &lambda_k);	// use error to save norm(A|p_k>)
		lambda_k.x = pow(error,2.)/lambda_k.x;
		lambda_k.y = 0.0;
		// calculate |x_(k+1)>
		hipblasZaxpy(handle, dimension, &lambda_k, p_k_dev, 1, x_k_dev, 1);
		// calcualte |r_(k+1)>
		lambda_k.x *= -1.0;
		hipblasZaxpy(handle, dimension, &lambda_k, b_dev, 1, r_k_dev, 1);
		// calculate u_k
		hipblasZdotc(handle, dimension, r_k_dev, 1, r_k_dev, 1, &u_k);
		u_k.x = u_k.x/pow(error,2.);
		u_k.y = 0.0;
		// calcualte |p_(k+1)>
		hipblasZscal(handle, dimension, &u_k, p_k_dev, 1.);
		hipblasZaxpy(handle, dimension, &alpha, r_k_dev, 1, p_k_dev, 1);
		// calculate error
		hipblasDznrm2(handle, dimension, r_k_dev, 1, &error);
		error /= norm;

		iter += 1;
//		printf("Iteation = %d ; error = %.16e .\n", iter, error);
	}
	printf("Iteation = %d ; error = %.16e .\n", iter, error);

	// get answer
	hipblasGetVector(dimension, sizeof(hipDoubleComplex), x_k_dev, 1, x_k_host, 1);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_GPU, start, stop);
	printf("\nGPU total computation time is %.2f ms.\n", time_GPU);

	scanf("%c", &print_out);
	puts("Print solution or not? (y/n)");
	scanf("%c", &print_out);
	if (print_out=='y')
	{
		printf("\nThe GPU solution vector is:\n");
		for (int i=0; i<dimension; i++)
			printf("%.16f%+.16fI\n", x_k_host[i].x, x_k_host[i].y);
		printf("\n");
	}
	else if (print_out!='n')
	{
		puts("Wrong input! Exit!");
		return EXIT_FAILURE;
	}
	// chcek answer
	puts("Do error check for GPU...");
	beta = make_hipDoubleComplex(-1.0, 0.0);
	hipblasSetVector(dimension, sizeof(hipDoubleComplex), b, 1, b_dev, 1);
	hipblasDznrm2(handle, dimension, b_dev, 1, &norm);
	hipblasSetMatrix(dimension, dimension, sizeof(hipDoubleComplex), A, dimension, A_dev, dimension);
	hipblasZgemv(handle, HIPBLAS_OP_N, dimension, dimension, &alpha, A_dev, dimension, x_k_dev, 1, &beta, b_dev, 1);
	error = 0.0;
	hipblasDznrm2(handle, dimension, b_dev, 1, &error);
	hipMemcpy(r_k_dev, b_dev, dimension*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	printf("The error for GPU is %.16e .\n", error/norm);

	hipblasDestroy(handle);
	hipFree(A_dev);
	hipFree(b_dev);
	hipFree(x_k_dev);
	hipFree(r_k_dev);
	hipFree(p_k_dev);
	
#if (DO_CPU_CHECK==true)
	puts("\nStart CPU comparsion...");
	hipDoubleComplex temp_scalar;
	hipDoubleComplex *r_k_host, *p_k_host, *temp_vector;
	hipEventRecord(start,0);

	x_k_host = (hipDoubleComplex *)calloc(dimension, sizeof(hipDoubleComplex));
	r_k_host = (hipDoubleComplex *)malloc(dimension*sizeof(hipDoubleComplex));
	p_k_host = (hipDoubleComplex *)malloc(dimension*sizeof(hipDoubleComplex));
	temp_vector = (hipDoubleComplex *)malloc(dimension*sizeof(hipDoubleComplex));

	//convert |b> to A^H|b>
	memcpy(temp_vector, b, dimension*sizeof(hipDoubleComplex));
	MV_PRODUCT('C', dimension, A, b, b);
	//conver A to A^HA

	memcpy(r_k_host, b, dimension*sizeof(hipDoubleComplex));
	memcpy(p_k_host, r_k_host, dimension*sizeof(hipDoubleComplex));
	norm = sqrt((VV_PRODUCT(dimension, b, b)).x);
	iter = 0;
	error = 1.0;
		
	while (error>criteria)
	{
		MV_PRODUCT('N', dimension, temp_matrix, p_k_host, b);
		temp_scalar = VV_PRODUCT(dimension, p_k_host, b);
		error = (VV_PRODUCT(dimension, r_k_host, r_k_host)).x;
		lambda_k.x = error/temp_scalar.x;
		lambda_k.y = 0.0;
		VV_ADDITION(dimension, &alpha, &lambda_k, x_k_host, p_k_host, x_k_host);
		lambda_k.x *= -1.0;
		VV_ADDITION(dimension, &alpha, &lambda_k, r_k_host, b, r_k_host);
			
		temp_scalar = VV_PRODUCT(dimension, r_k_host, r_k_host);
		u_k.x = temp_scalar.x/error;
		u_k.y = 0.0;
		VV_ADDITION(dimension, &alpha, &u_k, r_k_host, p_k_host, p_k_host);
		
		error = sqrt((VV_PRODUCT(dimension, r_k_host, r_k_host)).x)/norm;
		iter += 1;
		
//		printf("Iteation = %d ; error = %.16e .\n", iter, error);
	}
	printf("Iteation = %d ; error = %.16e .\n", iter, error);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_CPU, start, stop);
	printf("CPU total computation time is %.2f ms.\n", time_CPU);
	printf("The speed up is %.4f .\n", time_CPU/time_GPU);

	if (print_out=='y')
	{
		printf("\nThe CPU solution vector is:\n");
		for (int i=0; i<dimension; i++)
			printf("%.16f%+.16fI\n", x_k_host[i].x, x_k_host[i].y);
		printf("\n");
	}
	else if (print_out!='n')
	{
		puts("Wrong input! Exit!");
		return EXIT_FAILURE;
	}

	puts("Do error check for CPU...");
	beta = make_hipDoubleComplex(-1.0, 0.0);
	norm = sqrt((VV_PRODUCT(dimension, temp_vector, temp_vector)).x);
	MV_PRODUCT('N', dimension, A, x_k_host, b);	
	VV_ADDITION(dimension, &alpha, &beta, b, temp_vector, temp_vector);	// original |b> is saved as |temp_vector>
	error = sqrt((VV_PRODUCT(dimension, temp_vector, temp_vector)).x);
	printf("The error for CPU is %.16e .\n", error/norm);
	
	free(r_k_host);
	free(p_k_host);
	free(temp_vector);
#endif

	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(temp_matrix);
	free(x_k_host);
	free(A);
	free(b);
	return EXIT_SUCCESS;
}

void MHM_PRODUCT(int N, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *ANS)
{
	int index, index_dummy_A, index_dummy_B;
	hipDoubleComplex *temp = (hipDoubleComplex *)calloc(N*N, sizeof(hipDoubleComplex));
	for (int i=0; i<N; i++)
	{
		for (int j=0; j<N; j++)
		{
			index = IDX2C(j,i,N);
			for (int k=0; k<N; k++)
			{
				index_dummy_A = IDX2C(k,j,dimension);
				index_dummy_B = IDX2C(k,i,dimension);
				temp[index].x += A[index_dummy_A].x*B[index_dummy_B].x + A[index_dummy_A].y*B[index_dummy_B].y;
				temp[index].y += A[index_dummy_A].x*B[index_dummy_B].y - A[index_dummy_A].y*B[index_dummy_B].x;
			}
		}
	}
	memcpy(ANS, temp, N*N*sizeof(hipDoubleComplex));
	free(temp);
}

#if (DO_CPU_CHECK==true)
void VV_ADDITION(int N, hipDoubleComplex *alpha, hipDoubleComplex *beta, hipDoubleComplex *a, hipDoubleComplex* b, hipDoubleComplex* ans)
{
	hipDoubleComplex temp;
	for (int i=0; i<N; i++)
	{
		temp.x = ((*alpha).x*a[i].x - (*alpha).y*a[i].y) + ((*beta).x*b[i].x - (*beta).y*b[i].y);
		temp.y = ((*alpha).x*a[i].y + (*alpha).y*a[i].x) + ((*beta).x*b[i].y + (*beta).y*b[i].x);

		ans[i].x = temp.x;
		ans[i].y = temp.y;
	}
}

hipDoubleComplex VV_PRODUCT(int N, hipDoubleComplex *a, hipDoubleComplex *b)
{
	hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
	for (int i=0; i<N; i++)
	{
		sum.x += a[i].x*b[i].x + a[i].y*b[i].y;
		sum.y += a[i].x*b[i].y - a[i].y*b[i].x;
	}
	return(sum);
}

void MV_PRODUCT(char operation, int N, hipDoubleComplex *A, hipDoubleComplex *b, hipDoubleComplex *ans)
{
	int index;
	hipDoubleComplex *temp = (hipDoubleComplex *)calloc(N, sizeof(hipDoubleComplex));
	if (operation=='N')
	{
		for (int i=0; i<N; i++)
		{
			for (int j=0; j<N; j++)
			{
				index = IDX2C(i,j,dimension);
				temp[i].x += A[index].x*b[j].x - A[index].y*b[j].y;
				temp[i].y += A[index].x*b[j].y + A[index].y*b[j].x;
			}
		}
	}
	else if (operation=='C')
	{
		for (int i=0; i<N; i++)
		{
			for (int j=0; j<N; j++)
			{
				index = IDX2C(j,i,dimension);
				temp[i].x += A[index].x*b[j].x + A[index].y*b[j].y;
				temp[i].y += A[index].x*b[j].y - A[index].y*b[j].x;
			}
		}
	}
	else 
	{
		puts("Wrong input! Exit!");
		exit(1);
	}
	memcpy(ans, temp, N*sizeof(hipDoubleComplex));
	free(temp);
}
#endif
